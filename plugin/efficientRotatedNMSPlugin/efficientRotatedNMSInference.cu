#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common/bboxUtils.h"
#include "hipcub/hipcub.hpp"
#include "hip/hip_runtime_api.h"

#include "efficientRotatedNMSInference.cuh"
#include "efficientRotatedNMSInference.h"

#define NMS_TILES 5

using namespace nvinfer1;
using namespace nvinfer1::plugin;

template <typename T>
__device__ float IOU(EfficientRotatedNMSParameters param, RotatedBoxCorner<T> box1, RotatedBoxCorner<T> box2)
{
    // Regardless of the selected box coding, IOU is always performed in RotatedBoxCorner coding.
    // The boxes are copied so that they can be reordered without affecting the originals.
    RotatedBoxCorner<T> b1 = box1;
    RotatedBoxCorner<T> b2 = box2;
    b1.reorder();
    b2.reorder();
    return RotatedBoxCorner<T>::probiou(b1, b2);
}

template <typename T, typename Tb>
__device__ RotatedBoxCorner<T> DecodeBoxes(EfficientRotatedNMSParameters param, int boxIdx, int anchorIdx,
    const Tb* __restrict__ boxesInput, const Tb* __restrict__ anchorsInput)
{
    // The inputs will be in the selected coding format, as well as the decoding function. But the decoded box
    // will always be returned as RotatedBoxCorner.
    Tb box = boxesInput[boxIdx];
    if (!param.boxDecoder)
    {
        return RotatedBoxCorner<T>(box);
    }
    Tb anchor = anchorsInput[anchorIdx];
    box.reorder();
    anchor.reorder();
    return RotatedBoxCorner<T>(box.decode(anchor));
}

template <typename T, typename Tb>
__device__ void MapRotatedNMSData(EfficientRotatedNMSParameters param, int idx, int imageIdx, const Tb* __restrict__ boxesInput,
    const Tb* __restrict__ anchorsInput, const int* __restrict__ topClassData, const int* __restrict__ topAnchorsData,
    const int* __restrict__ topNumData, const T* __restrict__ sortedScoresData, const int* __restrict__ sortedIndexData,
    T& scoreMap, int& classMap, RotatedBoxCorner<T>& boxMap, int& boxIdxMap)
{
    // idx: Holds the NMS box index, within the current batch.
    // idxSort: Holds the batched NMS box index, which indexes the (filtered, but sorted) score buffer.
    // scoreMap: Holds the score that corresponds to the indexed box being processed by NMS.
    if (idx >= topNumData[imageIdx])
    {
        return;
    }
    int idxSort = imageIdx * param.numScoreElements + idx;
    scoreMap = sortedScoresData[idxSort];

    // idxMap: Holds the re-mapped index, which indexes the (filtered, but unsorted) buffers.
    // classMap: Holds the class that corresponds to the idx'th sorted score being processed by NMS.
    // anchorMap: Holds the anchor that corresponds to the idx'th sorted score being processed by NMS.
    int idxMap = imageIdx * param.numScoreElements + sortedIndexData[idxSort];
    classMap = topClassData[idxMap];
    int anchorMap = topAnchorsData[idxMap];

    // boxIdxMap: Holds the re-re-mapped index, which indexes the (unfiltered, and unsorted) boxes input buffer.
    boxIdxMap = -1;
    if (param.shareLocation) // Shape of boxesInput: [batchSize, numAnchors, 1, 4]
    {
        boxIdxMap = imageIdx * param.numAnchors + anchorMap;
    }
    else // Shape of boxesInput: [batchSize, numAnchors, numClasses, 4]
    {
        int batchOffset = imageIdx * param.numAnchors * param.numClasses;
        int anchorOffset = anchorMap * param.numClasses;
        boxIdxMap = batchOffset + anchorOffset + classMap;
    }
    // anchorIdxMap: Holds the re-re-mapped index, which indexes the (unfiltered, and unsorted) anchors input buffer.
    int anchorIdxMap = -1;
    if (param.shareAnchors) // Shape of anchorsInput: [1, numAnchors, 4]
    {
        anchorIdxMap = anchorMap;
    }
    else // Shape of anchorsInput: [batchSize, numAnchors, 4]
    {
        anchorIdxMap = imageIdx * param.numAnchors + anchorMap;
    }
    // boxMap: Holds the box that corresponds to the idx'th sorted score being processed by NMS.
    boxMap = DecodeBoxes<T, Tb>(param, boxIdxMap, anchorIdxMap, boxesInput, anchorsInput);
}

template <typename T>
__device__ void WriteRotatedNMSResult(EfficientRotatedNMSParameters param, int* __restrict__ numDetectionsOutput,
    T* __restrict__ nmsScoresOutput, int* __restrict__ nmsClassesOutput, RotatedBoxCorner<T>* __restrict__ nmsBoxesOutput,
    T threadScore, int threadClass, RotatedBoxCorner<T> threadBox, int imageIdx, unsigned int resultsCounter)
{
    int outputIdx = imageIdx * param.numOutputBoxes + resultsCounter - 1;
    if (param.scoreSigmoid)
    {
        nmsScoresOutput[outputIdx] = sigmoid_mp(threadScore);
    }
    else if (param.scoreBits > 0)
    {
        nmsScoresOutput[outputIdx] = add_mp(threadScore, (T) -1);
    }
    else
    {
        nmsScoresOutput[outputIdx] = threadScore;
    }
    nmsClassesOutput[outputIdx] = threadClass;
    if (param.clipBoxes)
    {
        nmsBoxesOutput[outputIdx] = threadBox.clip((T) 0, (T) 1);
    }
    else
    {
        nmsBoxesOutput[outputIdx] = threadBox;
    }
    numDetectionsOutput[imageIdx] = resultsCounter;
}

template <typename T, typename Tb>
__global__ void EfficientRotatedNMS(EfficientRotatedNMSParameters param, const int* topNumData, int* outputIndexData,
    int* outputClassData, const int* sortedIndexData, const T* __restrict__ sortedScoresData,
    const int* __restrict__ topClassData, const int* __restrict__ topAnchorsData, const Tb* __restrict__ boxesInput,
    const Tb* __restrict__ anchorsInput, int* __restrict__ numDetectionsOutput, T* __restrict__ nmsScoresOutput,
    int* __restrict__ nmsClassesOutput, RotatedBoxCorner<T>* __restrict__ nmsBoxesOutput)
{
    unsigned int thread = threadIdx.x;  ///< 线程 id
    unsigned int imageIdx = blockIdx.y; ///< 图像批次索引
    unsigned int tileSize = blockDim.x; ///< 线程块大小

    /// 检查图像索引有效性
    if (imageIdx >= param.batchSize)
    {
        return;
    }

    /// 确定实际处理的候选框数量
    int numSelectedBoxes = min(topNumData[imageIdx], param.numSelectedBoxes);
    /// 计算需要的分块数
    int numTiles = (numSelectedBoxes + tileSize - 1) / tileSize;
    /// 检查线程有效性
    if (thread >= numSelectedBoxes)
    {
        return;
    }

    __shared__ int blockState; ///< 块状态（线程间通信）
    __shared__ unsigned int resultsCounter; ///< 结果计数器
    if (thread == 0)
    {
        blockState = 0; ///< 初始状态; 0=正常
        resultsCounter = 0; ///< 结果计数清零
    }

    int threadState[NMS_TILES]; ///< 框状态
    unsigned int boxIdx[NMS_TILES]; ///< 框全局索引
    T threadScore[NMS_TILES]; ///< 框得分
    int threadClass[NMS_TILES]; ///< 框类别
    RotatedBoxCorner<T> threadBox[NMS_TILES]; ///< 旋转框索引
    int boxIdxMap[NMS_TILES]; ///< 框映射索引

    /// 初始化并加载数据
    for (int tile = 0; tile < numTiles; tile++)
    {
        threadState[tile] = 0; ///< 初始状态
        boxIdx[tile] = thread + tile * blockDim.x; ///< 计算全局索引
        /// 加载框数据到线程私有存储
        MapRotatedNMSData<T, Tb>(param, boxIdx[tile], imageIdx, boxesInput, anchorsInput, topClassData, topAnchorsData,
            topNumData, sortedScoresData, sortedIndexData, threadScore[tile], threadClass[tile], threadBox[tile],
            boxIdxMap[tile]);
    }

    // Iterate through all boxes to NMS against.
    for (int i = 0; i < numSelectedBoxes; i++)
    {
        int tile = i / tileSize; /// 当前框所属分块
        
        if (boxIdx[tile] == i)
        {
            // Iteration lead thread, figure out what the other threads should do,
            // this will be signaled via the blockState shared variable.
            if (threadState[tile] == -1)
            {
                // Thread already dead, this box was already dropped in a previous iteration,
                // because it had a large IOU overlap with another lead thread previously, so
                // it would never be kept anyway, therefore it can safely be skip all IOU operations
                // in this iteration.
                /// 状态：跳过迭代，框已被丢弃
                blockState = -1; // -1 => Signal all threads to skip iteration
            }
            else if (threadState[tile] == 0)
            {
                // As this box will be kept, this is a good place to find what index in the results buffer it
                // should have, as this allows to perform an early loop exit if there are enough results.
                if (resultsCounter >= param.numOutputBoxes)
                {
                    /// 状态：提前退出，结果数已达上限
                    blockState = -2; // -2 => Signal all threads to do an early loop exit.
                }
                else
                {
                    // Thread is still alive, because it has not had a large enough IOU overlap with
                    // any other kept box previously. Therefore, this box will be kept for sure. However,
                    // we need to check against all other subsequent boxes from this position onward,
                    // to see how those other boxes will behave in future iterations.
                    blockState = 1;        // +1 => Signal all (higher index) threads to calculate IOU against this box
                    threadState[tile] = 1; // +1 => Mark this box's thread to be kept and written out to results

                    // If the numOutputBoxesPerClass check is enabled, write the result only if the limit for this
                    // class on this image has not been reached yet. Other than (possibly) skipping the write, this
                    // won't affect anything else in the NMS threading.
                    bool write = true;
                    if (param.numOutputBoxesPerClass >= 0)
                    {
                        int classCounterIdx = imageIdx * param.numClasses + threadClass[tile];
                        write = (outputClassData[classCounterIdx] < param.numOutputBoxesPerClass);
                        outputClassData[classCounterIdx]++;
                    }
                    if (write)
                    {
                        // This branch is visited by one thread per iteration, so it's safe to do non-atomic increments.
                        resultsCounter++;
                        WriteRotatedNMSResult<T>(param, numDetectionsOutput, nmsScoresOutput, nmsClassesOutput,
                            nmsBoxesOutput, threadScore[tile], threadClass[tile], threadBox[tile], imageIdx,
                            resultsCounter);
                    }
                }
            }
            else
            {
                // This state should never be reached, but just in case...
                blockState = 0; // 0 => Signal all threads to not do any updates, nothing happens.
            }
        }

        __syncthreads();

        if (blockState == -2)
        {
            // This is the signal to exit from the loop.
            return; ///< 提取推出
        }

        if (blockState == -1)
        {
            // This is the signal for all threads to just skip this iteration, as no IOU's need to be checked.
            continue; ///< 跳过本次迭代
        }

        // Grab a box and class to test the current box against. The test box corresponds to iteration i,
        // therefore it will have a lower index than the current thread box, and will therefore have a higher score
        // than the current box because it's located "before" in the sorted score list.
        T testScore;
        int testClass;
        RotatedBoxCorner<T> testBox;
        int testBoxIdxMap;

        /// 获取测试框数据
        MapRotatedNMSData<T, Tb>(param, i, imageIdx, boxesInput, anchorsInput, topClassData, topAnchorsData, topNumData,
            sortedScoresData, sortedIndexData, testScore, testClass, testBox, testBoxIdxMap);

        for (int tile = 0; tile < numTiles; tile++)
        {
            bool ignoreClass = true;
            if (!param.classAgnostic)
            {
                ignoreClass = threadClass[tile] == testClass;
            }

            // IOU
            if (boxIdx[tile] > i && // 只处理索引更大的框 Make sure two different boxes are being tested, and that it's a higher index;
                boxIdx[tile] < numSelectedBoxes && // 索引有效 Make sure the box is within numSelectedBoxes;
                blockState == 1 &&                 // 状态允许计算 Signal that allows IOU checks to be performed;
                threadState[tile] == 0 &&          // 框状态为待处理 Make sure this box hasn't been either dropped or kept already;
                ignoreClass &&                     // 类别无关 Compare only boxes of matching classes when classAgnostic is false;
                lte_mp(threadScore[tile], testScore) && // 得分排序 Make sure the sorting order of scores is as expected;
                IOU<T>(param, threadBox[tile], testBox) >= param.iouThreshold) // IOU阈值判断 And... IOU overlap.
            {
                // Current box overlaps with the box tested in this iteration, this box will be skipped.
                threadState[tile] = -1; // -1 => Mark this box's thread to be dropped.
            }
        }
    }
}

template <typename T>
hipError_t EfficientRotatedNMSLauncher(EfficientRotatedNMSParameters& param, int* topNumData, int* outputIndexData,
    int* outputClassData, int* sortedIndexData, T* sortedScoresData, int* topClassData, int* topAnchorsData,
    const void* boxesInput, const void* anchorsInput, int* numDetectionsOutput, T* nmsScoresOutput,
    int* nmsClassesOutput, void* nmsBoxesOutput, hipStream_t stream)
{
    /// 根据候选框数量调整线程块大小，优化并行效率
    unsigned int tileSize = param.numSelectedBoxes / NMS_TILES;
    if (param.numSelectedBoxes <= 512)
    {
        tileSize = 512;
    }
    if (param.numSelectedBoxes <= 256)
    {
        tileSize = 256;
    }

    /// 每个线程块处理一个图像样本
    const dim3 blockSize = {tileSize, 1, 1};  ///< 一维线程块
    const dim3 gridSize = {1, (unsigned int) param.batchSize, 1}; ///< 二维网格

    if (param.boxCoding == 0)
    {
        /// 角点编码：x1 y1 x2 y2 r
        EfficientRotatedNMS<T, RotatedBoxCorner<T>><<<gridSize, blockSize, 0, stream>>>(param, topNumData, outputIndexData,
            outputClassData, sortedIndexData, sortedScoresData, topClassData, topAnchorsData,
            (RotatedBoxCorner<T>*) boxesInput, (RotatedBoxCorner<T>*) anchorsInput, numDetectionsOutput, nmsScoresOutput,
            nmsClassesOutput, (RotatedBoxCorner<T>*) nmsBoxesOutput);
    }
    else if (param.boxCoding == 1)
    {
        /// 中心编码：x y w h r
        // Note that nmsBoxesOutput is always coded as RotatedBoxCorner<T>, regardless of the input coding type.
        EfficientRotatedNMS<T, RotatedBoxCenterSize<T>><<<gridSize, blockSize, 0, stream>>>(param, topNumData, outputIndexData,
            outputClassData, sortedIndexData, sortedScoresData, topClassData, topAnchorsData,
            (RotatedBoxCenterSize<T>*) boxesInput, (RotatedBoxCenterSize<T>*) anchorsInput, numDetectionsOutput, nmsScoresOutput,
            nmsClassesOutput, (RotatedBoxCorner<T>*) nmsBoxesOutput);
    }

    return hipGetLastError();
}

__global__ void EfficientRotatedNMSFilterSegments(EfficientRotatedNMSParameters param, const int* __restrict__ topNumData,
    int* __restrict__ topOffsetsStartData, int* __restrict__ topOffsetsEndData)
{
    /// 每个线程处理一个图像样本 <<<1, param.batchSize, 0, stream>>>
    int imageIdx = threadIdx.x;
    if (imageIdx > param.batchSize)
    {
        return;
    }
    /// 当前图像在全局数组中的起始索引 图像索引*最大候选框数
    topOffsetsStartData[imageIdx] = imageIdx * param.numScoreElements;
    /// 当前图像在全局数组中的结束索引 起始索引+当前图像实际候选框数
    topOffsetsEndData[imageIdx] = imageIdx * param.numScoreElements + topNumData[imageIdx];
}

template <typename T>
__global__ void EfficientRotatedNMSFilter(EfficientRotatedNMSParameters param, const T* __restrict__ scoresInput,
    int* __restrict__ topNumData, int* __restrict__ topIndexData, int* __restrict__ topAnchorsData,
    T* __restrict__ topScoresData, int* __restrict__ topClassData)
{
    /// 候选框元素索引
    int elementIdx = blockDim.x * blockIdx.x + threadIdx.x;
    /// 图像批次索引
    int imageIdx = blockDim.y * blockIdx.y + threadIdx.y;

    // Boundary Conditions 边界检查
    if (elementIdx >= param.numScoreElements || imageIdx >= param.batchSize)
    {
        return;
    }

    // Shape of scoresInput: [batchSize, numAnchors, numClasses]
    /// 获取 num_classes 索引
    int scoresInputIdx = imageIdx * param.numScoreElements + elementIdx;

    // For each class, check its corresponding score if it crosses the threshold, and if so select this anchor,
    // and keep track of the maximum score and the corresponding (argmax) class id
    T score = scoresInput[scoresInputIdx];
    if (gte_mp(score, (T) param.scoreThreshold))
    {
        // Unpack the class and anchor index from the element index
        int classIdx = elementIdx % param.numClasses;  ///< 类别索引
        int anchorIdx = elementIdx / param.numClasses; ///< 目标框索引

        // If this is a background class, ignore it. 背景过滤
        if (classIdx == param.backgroundClass)
        {
            return;
        }

        // Use an atomic to find an open slot where to write the selected anchor data.
        /// 预检查：避免已满图像的无效原子操作
        if (topNumData[imageIdx] >= param.numScoreElements)
        {
            return;
        }
        int selectedIdx = atomicAdd((unsigned int*) &topNumData[imageIdx], 1);
        /// 后检查：确保写入位置不越界
        if (selectedIdx >= param.numScoreElements)
        {
            topNumData[imageIdx] = param.numScoreElements;
            return;
        }

        // Shape of topScoresData / topClassData: [batchSize, numScoreElements]
        int topIdx = imageIdx * param.numScoreElements + selectedIdx;

        /// 为后续的浮点数位排序做准备
        if (param.scoreBits > 0)
        {
            score = add_mp(score, (T) 1);
            if (gt_mp(score, (T) (2.f - 1.f / 1024.f)))
            {
                // Ensure the incremented score fits in the mantissa without changing the exponent
                score = (2.f - 1.f / 1024.f);
            }
        }

        topIndexData[topIdx] = selectedIdx; ///< 输出缓冲区索引
        topAnchorsData[topIdx] = anchorIdx; ///< 目标狂索引
        topScoresData[topIdx] = score;      ///< 优化后的分数
        topClassData[topIdx] = classIdx;    ///< 类别索引
    }
}

template <typename T>
__global__ void EfficientRotatedNMSDenseIndex(EfficientRotatedNMSParameters param, int* __restrict__ topNumData,
    int* __restrict__ topIndexData, int* __restrict__ topAnchorsData, int* __restrict__ topOffsetsStartData,
    int* __restrict__ topOffsetsEndData, T* __restrict__ topScoresData, int* __restrict__ topClassData)
{
    int elementIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int imageIdx = blockDim.y * blockIdx.y + threadIdx.y;

    if (elementIdx >= param.numScoreElements || imageIdx >= param.batchSize)
    {
        return;
    }

    int dataIdx = imageIdx * param.numScoreElements + elementIdx;
    int anchorIdx = elementIdx / param.numClasses;
    int classIdx = elementIdx % param.numClasses;
    if (param.scoreBits > 0)
    {
        T score = topScoresData[dataIdx];
        if (lt_mp(score, (T) param.scoreThreshold))
        {
            score = (T) 1;
        }
        else if (classIdx == param.backgroundClass)
        {
            score = (T) 1;
        }
        else
        {
            score = add_mp(score, (T) 1);
            if (gt_mp(score, (T) (2.f - 1.f / 1024.f)))
            {
                // Ensure the incremented score fits in the mantissa without changing the exponent
                score = (2.f - 1.f / 1024.f);
            }
        }
        topScoresData[dataIdx] = score;
    }
    else
    {
        T score = topScoresData[dataIdx];
        if (lt_mp(score, (T) param.scoreThreshold))
        {
            topScoresData[dataIdx] = -(1 << 15);
        }
        else if (classIdx == param.backgroundClass)
        {
            topScoresData[dataIdx] = -(1 << 15);
        }
    }

    topIndexData[dataIdx] = elementIdx;
    topAnchorsData[dataIdx] = anchorIdx;
    topClassData[dataIdx] = classIdx;

    if (elementIdx == 0)
    {
        // Saturate counters
        topNumData[imageIdx] = param.numScoreElements;
        topOffsetsStartData[imageIdx] = imageIdx * param.numScoreElements;
        topOffsetsEndData[imageIdx] = (imageIdx + 1) * param.numScoreElements;
    }
}

/// @brief 筛选出符合阈值要求的候选框，并为后续排序和NMS准备数据
/// @tparam T 
/// @param param 参数列表
/// @param scoresInput 原始 scores 数据
/// @param topNumData 每个样本保留的候选框数量
/// @param topIndexData 保留候选框的索引
/// @param topAnchorsData 保留候选框对应的锚点索引
/// @param topOffsetsStartData 分段排序的起始索引
/// @param topOffsetsEndData 分段排序的结束索引
/// @param topScoresData 过滤后的得分
/// @param topClassData 候选框类别ID
/// @param stream 
/// @return 
template <typename T>
hipError_t EfficientRotatedNMSFilterLauncher(EfficientRotatedNMSParameters& param, const T* scoresInput, int* topNumData,
    int* topIndexData, int* topAnchorsData, int* topOffsetsStartData, int* topOffsetsEndData, T* topScoresData,
    int* topClassData, hipStream_t stream)
{
    /// X 维度：按候选框元素并行 每块 512 个元素
    /// Y 维度：按图像批次并行 每块 1 个样本
    const unsigned int elementsPerBlock = 512;
    const unsigned int imagesPerBlock = 1;
    const unsigned int elementBlocks = (param.numScoreElements + elementsPerBlock - 1) / elementsPerBlock;
    const unsigned int imageBlocks = (param.batchSize + imagesPerBlock - 1) / imagesPerBlock;
    const dim3 blockSize = {elementsPerBlock, imagesPerBlock, 1};
    const dim3 gridSize = {elementBlocks, imageBlocks, 1};

    /// 将阈值反向计算得到 Logit 空间（Sigmoid 反函数）
    float kernelSelectThreshold = 0.007f;
    if (param.scoreSigmoid)
    {
        // Inverse Sigmoid
        if (param.scoreThreshold <= 0.f)
        {
            param.scoreThreshold = -(1 << 15);
        }
        else
        {
            param.scoreThreshold = logf(param.scoreThreshold / (1.f - param.scoreThreshold));
        }
        kernelSelectThreshold = logf(kernelSelectThreshold / (1.f - kernelSelectThreshold));
        // Disable Score Bits Optimization
        param.scoreBits = -1;
    }

    /// 根据阈值选择高效执行路径
    if (param.scoreThreshold < kernelSelectThreshold)
    {
        /**
         * 低阈值路径，保留大部分候选框
         * 避免条件分支，用内存复制换取计算效率
        */
        // A full copy of the buffer is necessary because sorting will scramble the input data otherwise.
        /// 直接复制全部score
        PLUGIN_CHECK_CUDA(hipMemcpyAsync(topScoresData, scoresInput,
            param.batchSize * param.numScoreElements * sizeof(T), hipMemcpyDeviceToDevice, stream));
        /// 生成密集索引
        EfficientRotatedNMSDenseIndex<T><<<gridSize, blockSize, 0, stream>>>(param, topNumData, topIndexData, topAnchorsData,
            topOffsetsStartData, topOffsetsEndData, topScoresData, topClassData);
    }
    else
    {
        /// 通过原子操作压缩有效数据，减少后续处理量
        EfficientRotatedNMSFilter<T><<<gridSize, blockSize, 0, stream>>>(
            param, scoresInput, topNumData, topIndexData, topAnchorsData, topScoresData, topClassData);

        /// 计算每个图像（在批次中）在过滤后的候选框的起始和结束偏移量
        EfficientRotatedNMSFilterSegments<<<1, param.batchSize, 0, stream>>>(
            param, topNumData, topOffsetsStartData, topOffsetsEndData);
    }

    return hipGetLastError();
}

template <typename T>
size_t EfficientRotatedNMSSortWorkspaceSize(int batchSize, int numScoreElements)
{
    size_t sortedWorkspaceSize = 0;
    hipcub::DoubleBuffer<T> keysDB(nullptr, nullptr);
    hipcub::DoubleBuffer<int> valuesDB(nullptr, nullptr);
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, sortedWorkspaceSize, keysDB, valuesDB,
        numScoreElements, batchSize, (const int*) nullptr, (const int*) nullptr);
    return sortedWorkspaceSize;
}

size_t EfficientRotatedNMSWorkspaceSize(int batchSize, int numScoreElements, int numClasses, DataType datatype)
{
    size_t total = 0;
    const size_t align = 256;
    // Counters
    // 3 for Filtering
    // 1 for Output Indexing
    // C for Max per Class Limiting
    size_t size = (3 + 1 + numClasses) * batchSize * sizeof(int);
    total += size + (size % align ? align - (size % align) : 0);
    // Int Buffers
    for (int i = 0; i < 4; i++)
    {
        size = batchSize * numScoreElements * sizeof(int);
        total += size + (size % align ? align - (size % align) : 0);
    }
    // Float Buffers
    for (int i = 0; i < 2; i++)
    {
        size = batchSize * numScoreElements * dataTypeSize(datatype);
        total += size + (size % align ? align - (size % align) : 0);
    }
    // Sort Workspace
    if (datatype == DataType::kHALF)
    {
        size = EfficientRotatedNMSSortWorkspaceSize<__half>(batchSize, numScoreElements);
        total += size + (size % align ? align - (size % align) : 0);
    }
    else if (datatype == DataType::kFLOAT)
    {
        size = EfficientRotatedNMSSortWorkspaceSize<float>(batchSize, numScoreElements);
        total += size + (size % align ? align - (size % align) : 0);
    }

    return total;
}

template <typename T>
T* EfficientRotatedNMSWorkspace(void* workspace, size_t& offset, size_t elements)
{
    T* buffer = (T*) ((size_t) workspace + offset);
    size_t align = 256;
    size_t size = elements * sizeof(T);
    size_t sizeAligned = size + (size % align ? align - (size % align) : 0);
    offset += sizeAligned;
    return buffer;
}

template <typename T>
pluginStatus_t EfficientRotatedNMSDispatch(EfficientRotatedNMSParameters param, const void* boxesInput, const void* scoresInput,
    const void* anchorsInput, void* numDetectionsOutput, void* nmsBoxesOutput, void* nmsScoresOutput,
    void* nmsClassesOutput, void* workspace, hipStream_t stream)
{
    // Clear Outputs (not all elements will get overwritten by the kernels, so safer to clear everything out)
    /// 1. 将所有输出缓冲区初始化为0，确保未处理的元素不会残留无效数据
    CSC(hipMemsetAsync(numDetectionsOutput, 0x00, param.batchSize * sizeof(int), stream), STATUS_FAILURE);
    CSC(hipMemsetAsync(nmsScoresOutput, 0x00, param.batchSize * param.numOutputBoxes * sizeof(T), stream), STATUS_FAILURE);
    CSC(hipMemsetAsync(nmsBoxesOutput, 0x00, param.batchSize * param.numOutputBoxes * 5 * sizeof(T), stream), STATUS_FAILURE);
    CSC(hipMemsetAsync(nmsClassesOutput, 0x00, param.batchSize * param.numOutputBoxes * sizeof(int), stream), STATUS_FAILURE);

    // Empty Inputs
    /// 2. 若输入得分元素数量为0（无有效检测），直接返回成功
    if (param.numScoreElements < 1)
    {
        return STATUS_SUCCESS;
    }

    // Counters Workspace
    /// 3. 分配临时内存用于中间计算结果
    size_t workspaceOffset = 0; ///< 计算workspace后，offset会移动
    int countersTotalSize = (3 + 1 + param.numClasses) * param.batchSize;
    /// 每个batch保留的检测框数量
    int* topNumData = EfficientRotatedNMSWorkspace<int>(workspace, workspaceOffset, countersTotalSize);
    /// 记录每个batch在排序后的起始和结束索引
    int* topOffsetsStartData = topNumData + param.batchSize;
    int* topOffsetsEndData = topNumData + 2 * param.batchSize;
    /// 存储最终输出的索引和类别信息
    int* outputIndexData = topNumData + 3 * param.batchSize;
    int* outputClassData = topNumData + 4 * param.batchSize;
    CSC(hipMemsetAsync(topNumData, 0x00, countersTotalSize * sizeof(int), stream), STATUS_FAILURE);
    hipError_t status = hipGetLastError();
    CSC(status, STATUS_FAILURE);

    // Other Buffers Workspace
    /// 4. 分配排序与过滤缓冲区
    int* topIndexData
        = EfficientRotatedNMSWorkspace<int>(workspace, workspaceOffset, param.batchSize * param.numScoreElements);
    int* topClassData
        = EfficientRotatedNMSWorkspace<int>(workspace, workspaceOffset, param.batchSize * param.numScoreElements);
    int* topAnchorsData
        = EfficientRotatedNMSWorkspace<int>(workspace, workspaceOffset, param.batchSize * param.numScoreElements);
    int* sortedIndexData
        = EfficientRotatedNMSWorkspace<int>(workspace, workspaceOffset, param.batchSize * param.numScoreElements);
    T* topScoresData = EfficientRotatedNMSWorkspace<T>(workspace, workspaceOffset, param.batchSize * param.numScoreElements);
    T* sortedScoresData
        = EfficientRotatedNMSWorkspace<T>(workspace, workspaceOffset, param.batchSize * param.numScoreElements);
    size_t sortedWorkspaceSize = EfficientRotatedNMSSortWorkspaceSize<T>(param.batchSize, param.numScoreElements);
    char* sortedWorkspaceData = EfficientRotatedNMSWorkspace<char>(workspace, workspaceOffset, sortedWorkspaceSize);
    hipcub::DoubleBuffer<T> scoresDB(topScoresData, sortedScoresData);
    hipcub::DoubleBuffer<int> indexDB(topIndexData, sortedIndexData);

    // Kernels 核心处理流程

    /// 5.1 过滤低分候选框
    status = EfficientRotatedNMSFilterLauncher<T>(param, (T*) scoresInput, topNumData, topIndexData, topAnchorsData,
        topOffsetsStartData, topOffsetsEndData, topScoresData, topClassData, stream);
    CSC(status, STATUS_FAILURE);
    
    /// 5.2 分段排序
    status = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(sortedWorkspaceData, sortedWorkspaceSize, scoresDB,
        indexDB, param.batchSize * param.numScoreElements, param.batchSize, topOffsetsStartData, topOffsetsEndData,
        param.scoreBits > 0 ? (10 - param.scoreBits) : 0, param.scoreBits > 0 ? 10 : sizeof(T) * 8, stream);
    CSC(status, STATUS_FAILURE);

    /// 5.3 执行旋转NMS
    status = EfficientRotatedNMSLauncher<T>(param, topNumData, outputIndexData, outputClassData, indexDB.Current(),
        scoresDB.Current(), topClassData, topAnchorsData, boxesInput, anchorsInput, (int*) numDetectionsOutput,
        (T*) nmsScoresOutput, (int*) nmsClassesOutput, nmsBoxesOutput, stream);
    CSC(status, STATUS_FAILURE);

    return STATUS_SUCCESS;
}

pluginStatus_t EfficientRotatedNMSInference(EfficientRotatedNMSParameters param, const void* boxesInput, const void* scoresInput,
    const void* anchorsInput, void* numDetectionsOutput, void* nmsBoxesOutput, void* nmsScoresOutput,
    void* nmsClassesOutput, void* workspace, hipStream_t stream)
{
    if (param.datatype == DataType::kFLOAT)
    {
        /// FP32 通用推理
        param.scoreBits = -1;
        return EfficientRotatedNMSDispatch<float>(param, boxesInput, scoresInput, anchorsInput, numDetectionsOutput,
            nmsBoxesOutput, nmsScoresOutput, nmsClassesOutput, workspace, stream);
    }
    else if (param.datatype == DataType::kHALF)
    {
        /// FP16 推理
        if (param.scoreBits <= 0 || param.scoreBits > 10)
        {
            param.scoreBits = -1;
        }
        return EfficientRotatedNMSDispatch<__half>(param, boxesInput, scoresInput, anchorsInput, numDetectionsOutput,
            nmsBoxesOutput, nmsScoresOutput, nmsClassesOutput, workspace, stream);
    }
    else
    {
        return STATUS_NOT_SUPPORTED;
    }
}
